
#include <hip/hip_runtime.h>
__global__ void cudaTransientFstatExpWindow ( float *input,
                                              unsigned int numAtoms,
                                              unsigned int TAtom,
                                              unsigned int t0_data,
                                              unsigned int win_t0,
                                              unsigned int win_dt0,
                                              unsigned int win_tau,
                                              unsigned int win_dtau,
                                              unsigned int Fmn_rows,
                                              unsigned int Fmn_cols,
                                              float *Fmn
                                            )
{

  /* match CUDA thread indexing and high-level (t0,tau) indexing */
  unsigned int m         = blockDim.x * blockIdx.x + threadIdx.x; // t0:  row
  unsigned int n         = blockDim.y * blockIdx.y + threadIdx.y; // tau: column
  /* unraveled 1D index for 2D output array */
  unsigned int outidx    = Fmn_cols * m + n;

  /* hardcoded copy from lalpulsar */
  unsigned int TRANSIENT_EXP_EFOLDING = 3;

  if ( (m < Fmn_rows) && (n < Fmn_cols) ) {

    /* compute Fstat-atom index i_t0 in [0, numAtoms) */
    unsigned int TAtomHalf = TAtom/2; // integer division
    unsigned int t0 = win_t0 + m * win_dt0;
    /* integer round: floor(x+0.5) */
    int i_tmp = ( t0 - t0_data + TAtomHalf ) / TAtom;
    if ( i_tmp < 0 ) {
        i_tmp = 0;
    }
    unsigned int i_t0 = (unsigned int)i_tmp;
    if ( i_t0 >= numAtoms ) {
        i_t0 = numAtoms - 1;
    }

    /* translate n into an atoms end-index
     * for this search interval [t0, t0+Tcoh],
     * giving the index range of atoms to sum over
     */
    unsigned int tau = win_tau + n * win_dtau;

    /* get end-time t1 of this transient-window search
     * for given tau, what Tcoh should the exponential window cover?
     * for speed reasons we want to truncate
     * Tcoh = tau * TRANSIENT_EXP_EFOLDING
     * with the e-folding factor chosen such that the window-value
     * is practically negligible after that, where it will be set to 0
     */
//     unsigned int t1 = lround( win_t0 + TRANSIENT_EXP_EFOLDING * win_tau);
    unsigned int t1 = t0 + TRANSIENT_EXP_EFOLDING * tau;

      /* compute window end-time Fstat-atom index i_t1 in [0, numAtoms)
       * using integer round: floor(x+0.5)
       */
    i_tmp = ( t1 - t0_data + TAtomHalf ) / TAtom  - 1;
    if ( i_tmp < 0 ) {
        i_tmp = 0;
    }
    unsigned int i_t1 = (unsigned int)i_tmp;
    if ( i_t1 >= numAtoms ) {
        i_t1 = numAtoms - 1;
    }

    /* now we have two valid atoms-indices [i_t0, i_t1]
     * spanning our Fstat-window to sum over
     */

    float Ad    = 0.0f;
    float Bd    = 0.0f;
    float Cd    = 0.0f;
    float Fa_re = 0.0f;
    float Fa_im = 0.0f;
    float Fb_re = 0.0f;
    float Fb_im = 0.0f;

    unsigned short input_cols = 7; // must match input matrix!

    /* sum up atoms */
    for ( unsigned int i=i_t0; i<=i_t1; i++ ) {

      unsigned int t_i = t0_data + i * TAtom;

      float win_i = 0.0;
      if ( t_i >= t0 && t_i <= t1 ) {
        float x = 1.0 * ( t_i - t0 ) / tau;
        win_i = exp ( -x );
      }

      float win2_i = win_i * win_i;

      Ad    += input[i*input_cols+0] * win2_i; // a2_alpha
      Bd    += input[i*input_cols+1] * win2_i; // b2_alpha
      Cd    += input[i*input_cols+2] * win2_i; // ab_alpha
      Fa_re += input[i*input_cols+3] * win_i; // Fa_alpha_re
      Fa_im += input[i*input_cols+4] * win_i; // Fa_alpha_im
      Fb_re += input[i*input_cols+5] * win_i; // Fb_alpha_re
      Fb_im += input[i*input_cols+6] * win_i; // Fb_alpha_im

    }

    /* get determinant */
    float Dd = ( Ad * Bd - Cd * Cd );
    float DdInv = 0.0f;
    /* safety catch as in XLALWeightMultiAMCoeffs():
     * make it so that in the end F=0 instead of -nan
     */
    if ( Dd > 0.0 ) {
      DdInv  = 1.0 / Dd;
    }

    /* from XLALComputeFstatFromFaFb */
    float F  = DdInv * (  Bd * ( Fa_re*Fa_re + Fa_im*Fa_im )
                        + Ad * ( Fb_re*Fb_re + Fb_im*Fb_im )
                        - 2.0 * Cd * ( Fa_re * Fb_re + Fa_im * Fb_im )
                       );

    /* store result in Fstat-matrix
     * at unraveled index of element {m,n}
     */
    Fmn[outidx] = F;

  } // ( (m < Fmn_rows) && (n < Fmn_cols) )

} // cudaTransientFstatExpWindow()
